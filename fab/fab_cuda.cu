
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

__constant__ double params[4];

#define fabn(n) \
    ((long)(params[1] * (powf(params[2], n) - powf(params[3], n))))

__global__ static void fab(long *nums) {
    nums[threadIdx.x] = fabn((double)threadIdx.x + 3);
}

/***** Begin *****/
int main()
{
    int n;
    scanf("%d", &n);
    clock_t start = clock();
    if (n == 1)
        printf("1\n");
    else if (n == 2)
        printf("1 1\n");
    else {
        printf("1 1");
        n -= 2;
        double params_host[4] = {sqrt(5.0)};
        params_host[1] = 1 / params_host[0];
        params_host[2] = (1 + params_host[0]) / 2;
        params_host[3] = (1 - params_host[0]) / 2;
        hipMemcpyToSymbol(HIP_SYMBOL(params), params_host, sizeof(double) * 4);
        long* nums;
        hipMalloc(&nums, sizeof(long) * n);
        fab<<<1, n>>>(nums);
        hipDeviceSynchronize();
        long* nums_host = (long*)malloc(sizeof(long) * n);
        hipMemcpy(nums_host, nums, sizeof(long) * n, hipMemcpyDeviceToHost);
        hipFree(nums);
        hipFree(params);
        for (int i = 0; i < n; i++)
            printf(" %ld", nums_host[i]);
        printf("\n");
        free(nums_host);
    }
    printf("time: %lf\n", ((double)clock() - start) / CLOCKS_PER_SEC * 1000);
	return 0;
}
/***** End *****/